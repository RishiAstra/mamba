#include "hip/hip_runtime.h"
// mamba2_fused_cuda.cu
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <c10/util/Optional.h>
#include <hip/hip_fp16.h>
#include <tuple>

#define DIV_UP(a, b) (((a) + (b) - 1) / (b))

// Compile-time tuning bag (we'll keep CHUNK_SIZE unused for now; runtime param is passed)
template <uint32_t CHUNK_SIZE_, uint32_t THREADS_, uint32_t TARGET_BLOCKS_>
struct ssd_template_params {
    static constexpr uint32_t CHUNK_SIZE     = CHUNK_SIZE_;
    static constexpr uint32_t THREADS        = THREADS_;
    static constexpr uint32_t TARGET_BLOCKS  = TARGET_BLOCKS_;
    static_assert(THREADS % 32 == 0, "THREADS must be a multiple of 32");
};

// --- helpers ---
__device__ __forceinline__ float softplusf(float x) {
    // match triton's guard: skip exp for large x
    return (x <= 20.f) ? log1pf(expf(x)) : x;
}

// Step 1 kernel: (dt[, +bias][, softplus]) -> dt_out; dA = dt * A -> cumsum along chunk
// One block handles a single (batch b, chunk c, head h). Thread 0 does the serial scan
// (correctness-first; we can make this a parallel scan later).
template <typename Params>
__global__ __launch_bounds__(Params::THREADS, Params::TARGET_BLOCKS)
void mamba2_ssd_step1_dt_transform_cumsum_kernel(
    const __half* __restrict__ dt,     // [B, S, H] contiguous
    const float*  __restrict__ A,      // [H]
    const __half* __restrict__ dt_bias,// [H]
    int B, int S, int H,
    int chunk_size,
    bool dt_softplus,
    float* __restrict__ dt_out,        // [B, H, n_chunks, chunk_size] fp32
    float* __restrict__ dA_cumsum      // same shape
) {
    const int b = blockIdx.y;
    const int chunk_id = blockIdx.x;
    const int h = blockIdx.z;

    if (b >= B || h >= H) return;

    const int s_start = chunk_id * chunk_size;
    if (s_start >= S) return;
    const int this_chunk_len = min(chunk_size, S - s_start);

    const int n_chunks = DIV_UP(S, chunk_size);

    // gather scalars per head
    const float A_h = A[h];
    const float bias_h = __half2float(dt_bias[h]);

    // TODO: support chunk size != block size
    const int tid  = threadIdx.x;
    const int lane = tid & 31;
    const int warp_id = tid >> 5;
    constexpr int WARPS = Params::THREADS / 32;
    __shared__ float warp_sums[WARPS];

    // compute dt_val (masked for tail) and write dt_out
    float dt_val = 0.f;
    if (tid < this_chunk_len) {
        const int s = s_start + tid;
        const int64_t dt_idx = ((static_cast<int64_t>(b) * S + s) * H + h);
        dt_val = __half2float(dt[dt_idx]) + bias_h;
        if (dt_softplus) dt_val = softplusf(dt_val);
        // store dt_out[b,h,chunk,tid]
        const int64_t out_idx = ((static_cast<int64_t>(b) * H + h) * n_chunks + chunk_id) * chunk_size + tid;
        dt_out[out_idx] = dt_val;
    } else {
        // ensure padded tail is deterministic zero
        const int64_t out_idx = ((static_cast<int64_t>(b) * H + h) * n_chunks + chunk_id) * chunk_size + tid;
        if (tid < chunk_size) dt_out[out_idx] = 0.f;
    }

    // local value to scan
    float v = (tid < this_chunk_len) ? (dt_val * A_h) : 0.f;

    // intra-warp inclusive scan
    unsigned mask = 0xffffffffu;
    #pragma unroll
    for (int offset = 1; offset < 32; offset <<= 1) {
        float n = __shfl_up_sync(mask, v, offset);
        if (lane >= offset) v += n;
    }

    // TODO: fix for chunk_size != block_size
    // compute warp total robustly (handle partial last warp)
    const int warp_base = warp_id * 32;
    int local_count = this_chunk_len - warp_base;
    if (local_count < 0) local_count = 0;
    if (local_count > 32) local_count = 32;
    float warp_total = (local_count > 0) ? __shfl_sync(mask, v, local_count - 1) : 0.f;

    // one thread per warp stores total
    if (lane == 0) warp_sums[warp_id] = warp_total;
    __syncthreads();

    // small serial prefix over warps (W <= 8 here)
    if (threadIdx.x == 0) {
        float accw = 0.f;
        #pragma unroll
        for (int w = 0; w < WARPS; ++w) {
            float t = warp_sums[w];
            accw += t;
            warp_sums[w] = accw;  // inclusive
        }
    }
    __syncthreads();

    // add prefix of previous warps
    float warp_prefix = (warp_id == 0) ? 0.f : warp_sums[warp_id - 1];
    v += warp_prefix;

    // write dA_cumsum
    const int64_t out_idx = ((static_cast<int64_t>(b) * H + h) * n_chunks + chunk_id) * chunk_size + tid;
    if (tid < this_chunk_len) {
        dA_cumsum[out_idx] = v;
    } else if (tid < chunk_size) {
        dA_cumsum[out_idx] = 0.f;
    }
}


// ---------- public entry ----------
std::tuple<at::Tensor, at::Tensor> mamba2_fused_ssd_combined_fwd(
    const at::Tensor& x,          // [B, S, H, D] (unused in step1, we use it to infer B,S,H)
    const at::Tensor& dt,         // [B, S, H]   half
    const at::Tensor& A,          // [H]         float
    const at::Tensor& B,          // (unused here)
    const at::Tensor& C,          // (unused here)
    int64_t            chunk_size,
    const at::Tensor& D,          // (unused here)
    c10::optional<at::Tensor> z,  // (unused here)
    const at::Tensor& dt_bias,    // [H]         half
    c10::optional<at::Tensor> initial_states, // (unused here)
    c10::optional<at::Tensor> seq_idx,        // (unused here)
    c10::optional<at::Tensor> cu_seqlens,     // (unused here)
    bool dt_softplus
) {
    // --- device / dtype sanity ---
    auto dev = x.device();
    TORCH_CHECK(x.is_cuda(),        "x must be CUDA");
    TORCH_CHECK(dt.is_cuda(),       "dt must be CUDA");
    TORCH_CHECK(A.is_cuda(),        "A must be CUDA");
    TORCH_CHECK(dt_bias.is_cuda(),  "dt_bias must be CUDA");
    TORCH_CHECK(x.scalar_type() == at::kHalf, "x must be float16 (Half)");
    TORCH_CHECK(dt.scalar_type() == at::kHalf, "dt must be float16 (Half)");
    TORCH_CHECK(A.scalar_type()  == at::kFloat, "A must be float32");
    TORCH_CHECK(dt_bias.scalar_type() == at::kHalf, "dt_bias must be float16 (Half)");

    // --- infer shapes ---
    TORCH_CHECK(x.dim() >= 3, "x must have shape [B, S, H, ...]");
    const int Bsz = static_cast<int>(x.size(0));
    const int S   = static_cast<int>(x.size(1));
    const int H   = static_cast<int>(x.size(2));
    TORCH_CHECK(dt.sizes() == at::IntArrayRef({Bsz, S, H}), "dt must be [B,S,H]");
    TORCH_CHECK(A.sizes()  == at::IntArrayRef({H}),         "A must be [H]");
    TORCH_CHECK(dt_bias.sizes() == at::IntArrayRef({H}),    "dt_bias must be [H]");
    TORCH_CHECK(chunk_size > 0, "chunk_size must be > 0");

    const int n_chunks = DIV_UP(S, static_cast<int>(chunk_size));

    // --- allocate outputs for step 1 (fp32) ---
    auto opts_f = at::TensorOptions().dtype(at::kFloat).device(dev);
    auto dt_out    = at::empty({Bsz, H, n_chunks, static_cast<int>(chunk_size)}, opts_f);
    auto dA_cumsum = at::empty_like(dt_out);

    // --- launch kernel (correctness-first mapping: 1 head per block) ---
    using Params = ssd_template_params</*CHUNK_SIZE*/0, /*THREADS*/128, /*TARGET_BLOCKS*/4>;
    dim3 block(Params::THREADS);
    dim3 grid(n_chunks, Bsz, H);
    auto stream = at::cuda::getCurrentCUDAStream();

    mamba2_ssd_step1_dt_transform_cumsum_kernel<Params><<<grid, block, 0, stream>>>(
        reinterpret_cast<const __half*>(dt.data_ptr<at::Half>()),
        A.data_ptr<float>(),
        reinterpret_cast<const __half*>(dt_bias.data_ptr<at::Half>()),
        Bsz, S, H,
        static_cast<int>(chunk_size),
        dt_softplus,
        dt_out.data_ptr<float>(),
        dA_cumsum.data_ptr<float>()
    );

#ifndef NDEBUG
    auto err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA launch failed (step1): ", hipGetErrorString(err));
#endif

    // TODO: when we wire the rest of the fused pipeline, return a tuple:
    // (out, out_x, dt_out, dA_cumsum, states, final_states)
    // For now, keep API unchanged and return x (contiguous) to not break callers.
    // return x.contiguous();
    // Return exactly (dA_cumsum, dt_out) to match your Python expectation
    return std::make_tuple(dA_cumsum, dt_out);
}
